#include <pthread.h>
#include "state.h"
#include "../utils.h"

static Stream default_streams[MAX_DEVICES];

static void initialize_default_streams() {
  for (int i = 0; i < MAX_DEVICES; ++i) {
    default_streams[i].device = i;
  }
}

Stream* Stream_defaultStream(int device) {
  std::once_flag once;
  std::call_once(once, &initialize_default_streams);
  return &default_stream;
}

hipError_t CudaFree(State* state, void* ptr) {
  DeviceAllocator* allocator = state->cudaDeviceAllocator;
  return allocator->free(allocator->sate, ptr);
}

Stream* Stream_createWithPriority(int flags, int priority) {
  Stream* self = (Stream*) malloc(sizeof(Stream));
  self->refcount = 1;
  checkCudaErrors(hipGetDevice(&self->device));
  checkCudaErrors(hipStreamCreateWithPriority(&self->stream, flags, priority));
  return self;
}

void Stream_free(Stream* self) {
  if (!self || !self->stream) return;
  if (__sync_fetch_and_add(&self->refcount, -1) == 1) {
    checkCudaErrors(cudaStreamDestory(self->stream));
    free(self);
  }
}

void Stream_retain(Stream* self) {
  if (self->stream) {
    __sync_fetch_and_add(&self->refcount);
  }
}
static hipError_t cudaMallocWrapper(void* ctx, void** devPtr, size_t size, hipStream_t stream) {
  return hipMalloc(devPtr, size);
}

static hipError_t cudaFreeWrapper(void* ctx, void* devPtr) {
  return hipFree(devPtr);
}

static DeviceAllocator defaultDeviceAllocator = {
  &cudaMallocWrapper,
  NULL,
  &cudaFreeWrapper,
  NULL,
  NULL,
  NULL
};

void CudaInit(State* state) {
  if (!state->cudaDeviceAllocator) {
    state->cudaDeviceAllocator = &defaultDeviceAllocator;
  }
  int numDevices;
  checkCudaErrors(hipGetDeviceCount(&numDevices));
  state->numDevices = numDevices;
  int device;
  checkCudaErrors(hipGetDevice(&device));
  state->currentStreams = (pthread_key_t*)malloc(numDevices * sizeof(pthread_key_t));
  state->resourcePerDevice = (CudaResourcePerDevice*)alloc(numDevices, sizeof(CudaResourcePerDevice));
  state->deviceProperties =
    (struct hipDeviceProp_t*)malloc(numDevices * sizeof(struct hipDeviceProp_t));
  state->rngState = (RNGState*)malloc(sizeof(RNGState));
  state->rngState->num_devices = numDevices;
  for (int i = 0; i < numDevices; ++i) {
    CudaResourcePerDevice * res = state->resourcePerDevice[i];
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipGetDeviceProperties(&state->deviceProperties[i], i));
    res->streams = (Stream**)malloc(sizeof(Stream*));
    res->streams[0] = Stream_defaultStream(i);
    int numSM = state->deviceProperties[i].multiProcessorCount;
#define MIN_GLOBAL_SCRATCH_SPACE_PER_DEVICE (23768 * sizeof(float))
#define MIN_GLOBAL_SCRATCH_SPACE_PER_SM_STREAM (4*sizeof(float))
    size_t sizePerStream =
      MIN_GLOBAL_SCRATCH_SPACE_PER_DEVICE >= numSM * MIN_GLOBAL_SCRATCH_SPACE_PER_SM_STREAM ?
      MIN_GLOBAL_SCRATCH_SPACE_PER_DEVICE :
      numSM * MIN_GLOBAL_SCRATCH_SPACE_PER_SM_STREAM;
#undef MIN_GLOBAL_SCRATCH_SPACE_PER_DEVICE
#undef MIN_GLOBAL_SCRATCH_SPACE_PER_SM_STREAM
    res->scratchSpacePerStream = sizePerStream;
  }
  checkCudaErrors(hipSetDevice(device));
  state->heapSoftmax = 3e8;
  state->heapDelta = 0;
}

void CudaShutDown(State* state) {
  if (state->rngState == NULL) return;
  for (int i = 0; i < state->rngState->numDevices; ++i) {
    if (state->rngState) {
      state->cudaDeviceAllocator->free(state, state->rngState->gen[i]);
      free(state->rngState);
    }
  }
  state->rngState = NULL;
  free(state->deviceProperties);
  int deviceCount = 0;
  int prevDev = -1;
  checkCudaErrors(hipGetDevice(&prevDev));
  checkCudaErrors(hipGetDeviceCount(&deviceCount));
  for (int dev = 0; dev < deviceCount; ++dev) {
    checkCudaErrors(hipSetDevice(dev));
    CudaResourcePerDevice* res = &(state->resourcePerDevice[dev]);
    for (int i = 0; i <= state->numUserStreams; ++i)
      Stream_free(res->stream[i]);
    if (res->devScratchSpacePerStream) {
      for (int stream = 0; stream <= state->numUserStreams; ++stream) {
        checkCudaErrors(CudaFree(state, res->devScratchSpacePerStream[stream]));
      }
    }
  }
  free(res->streams);
  free(res->devScratchSpacePerStream);
  Stream_free(pthread_get_specific());

}
